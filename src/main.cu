#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   main.cu
 *  @author Thomas Müller, NVIDIA
 */

#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

#include <args/args.hxx>

#include <filesystem/path.h>

using namespace args;
using namespace ngp;
using namespace std;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

#define DEF_GUI_WIDTH "1920"
#define DEF_GUI_HEIGHT "1080"
#define DEF_MAX_TIME "36000" // 10 hours
#define DEF_MAX_PSNR "50.00"
#define DEF_MAX_EPOCH "1000000"

#define file_like(filename, likes) equals_case_insensitive(filename.extension(), likes)


inline float psnr(float x)
{
	return -10.f*logf(x)/logf(10.f);
}

void load_model(Testbed &testbed, fs::path &filename)
{
    if (! filename.exists()) {
		tlog::warning() << "Model file '" << filename.str() << "' does not exist";
		return;
    }

    if (! file_like(filename, "msgpack")) {
		tlog::warning() << "Model should be '*.msgpack' file";
		return;
    }

	testbed.load_snapshot(filename);
}

void save_model(Testbed &testbed, fs::path &filename)
{
	if (! file_like(filename, "msgpack")) {
		tlog::warning() << "Model should be '*.msgpack' file.";
		return;
	}
	// else
	testbed.save_snapshot(filename, false /*include_optimizer_state*/);
}


void save_mesh(Testbed &testbed, fs::path &filename)
{
    if (! file_like(filename, "obj")) {
		tlog::warning() << "Mesh should be *.obj file.";
		return;
    }

	if (testbed.m_testbed_mode != ETestbedMode::Nerf && 
		testbed.m_testbed_mode != ETestbedMode::Sdf) {
		tlog::warning() << "Save mesh only for NeRF or SDF.";
		return;
	}

	float thresh = (testbed.m_testbed_mode == ETestbedMode::Nerf)? 2.5f : 0.0f;
	// std::numeric_limits<float>::max()
	bool uv_flag = false; // file_like(filename, "obj"); /*generate_uvs_for_obj_file*/
	testbed.compute_and_save_marching_cubes_mesh(filename.str().c_str(),
		Eigen::Vector3i{256, 256, 256}, {} /*BoundingBox*/, thresh, uv_flag);
}

void save_point(Testbed &testbed, fs::path &filename)
{
    if (! file_like(filename, "png")) {
		tlog::warning() << "Point cloud should be *.ply file.";
		return;
    }

	if (testbed.m_testbed_mode != ETestbedMode::Nerf) {
		tlog::warning() << "Save point cloud only for NeRF.";
		return;
	}

	// testbed.get_nerf_rays_from_image(5);
	
	// testbed.save_nerf_point_cloud(filename.str().c_str());

	testbed.render_nerf_image(0, filename);
}


int main_func(const std::vector<std::string>& arguments) {
	ArgumentParser parser{
		"Instant Neural Graphics Primitives\n"
		"Version " NGP_VERSION,
		"",
	};

	HelpFlag help_flag{
		parser,
		"HELP",
		"Display this help.",
		{'h', "help"},
	};

	Flag version_flag{
		parser,
		"VERSION",
		"Display version.",
		{'v', "version"},
	};

	Flag no_gui_flag{
		parser,
		"NO_GUI",
		"Disable GUI.",
		{"no-gui"},
	};

	ValueFlag<uint32_t> width_flag{
		parser,
		DEF_GUI_WIDTH,
		"GUI width.",
		{"width"},
	};

	ValueFlag<uint32_t> height_flag{
		parser,
		DEF_GUI_HEIGHT,
		"GUI height.",
		{"height"},
	};

// #ifdef NGP_GUI
// 	Flag vr_flag{
// 		parser,
// 		"VR",
// 		"Enable VR",
// 		{"vr"}
// 	};
// #endif

	ValueFlag<string> load_config_flag{
		parser,
		"FILE_NAME",
		"Load net config from *.json file.",
		{"load_config"},
	};

	ValueFlag<string> load_model_flag{
		parser,
		"FILE_NAME",
		"Load model from *.msgpack file.",
		{"load_model"},
	};

	ValueFlag<string> save_model_flag{
		parser,
		"FILE_NAME",
		"Save model to *.msgpack file.",
		{"save_model"},
	};

	ValueFlag<string> save_mesh_flag{
		parser,
		"FILE_NAME",
		"Save mesh to *.obj file for NeRF or SDF.",
		{"save_mesh"},
	};

	ValueFlag<string> save_point_flag{
		parser,
		"FILE_NAME",
		"Save point cloud to *.ply file for NeRF.",
		{"save_point"},
	};

	ValueFlag<string> load_data_flag{
		parser,
		"DATASET",
		"Load training data from dataset (Folder for NeRF, *.obj/*.stl for SDF, *.nvdb for volume, others for image ).",
		{"load_data"},
	};

	Flag no_train_flag{
		parser,
		"NO_TRAIN",
		"Disable training.",
		{"no-train"},
	};

	ValueFlag<int32_t> max_epoch_flag{
		parser,
		DEF_MAX_EPOCH,
		"Training stop if epoch >= max_epoch.",
		{"max_epoch"},
	};

	ValueFlag<int32_t> max_time_flag{
		parser,
		DEF_MAX_TIME,
		"Training stop if time >= max_time seconds.",
		{"max_time"},
	};

	ValueFlag<float> max_psnr_flag{
		parser,
		DEF_MAX_PSNR,
		"Training stop if PSNR >= max_psnr.",
		{"max_psnr"},
	};

	PositionalList<string> files{
		parser,
		"files",
		"Files to be loaded. Can be a dataset, network config, snapshot, camera path, or a combination of those.",
	};

	// Parse command line arguments and react to parsing
	// errors using exceptions.
	try {
		if (arguments.empty()) {
			tlog::error() << "Argument number must be > 0.";
			return -3;
		}

		parser.Prog(arguments.front());
		parser.ParseArgs(begin(arguments) + 1, end(arguments));
	} catch (const Help&) {
		cout << parser;
		return 0;
	} catch (const ParseError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -1;
	} catch (const ValidationError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -2;
	}

	if (version_flag) {
		tlog::none() << "Instant Neural Graphics Primitives v" NGP_VERSION;
		return 0;
	}

	// Start ...
	Testbed testbed;

	for (auto file : get(files)) {
		testbed.load_file(file);
	}

	if (load_data_flag) {
		testbed.load_training_data(get(load_data_flag));
	}

	testbed.m_train = !no_train_flag;

	if (load_model_flag) {
	    fs::path filename = get(load_model_flag);
	    load_model(testbed, filename);
	}
	if (load_config_flag) {
		testbed.reload_network_from_file(get(load_config_flag));
	}


#ifdef NGP_GUI
	bool gui = !no_gui_flag;
#else
	bool gui = false;
#endif

	if (gui) {
		testbed.init_window(width_flag ? get(width_flag) : atoi(DEF_GUI_WIDTH),
			height_flag ? get(height_flag) : atoi(DEF_GUI_HEIGHT));
	}

// #ifdef NGP_GUI
// 	if (vr_flag) {
// 		testbed.init_vr();
// 	}
// #endif

	// Render/training loop
	float curr_psnr = 0.0f;
	std::time_t start_time = std::time(nullptr);
	float max_psnr = (max_psnr_flag)? get(max_psnr_flag) : atof(DEF_MAX_PSNR);
	uint32_t max_time = (max_time_flag)? get(max_time_flag) : atoi(DEF_MAX_TIME);
	uint32_t max_epoch = (max_epoch_flag)? get(max_epoch_flag) : atoi(DEF_MAX_EPOCH);

	testbed.redraw_gui_next_frame();
	while (testbed.frame()) {
		if (testbed.m_training_step % 100 != 0)
			continue;

		curr_psnr = psnr(testbed.m_loss_scalar.val());
		tlog::info() << "iteration=" << testbed.m_training_step 
				<< " loss=" << testbed.m_loss_scalar.val()
				<< " psnr=" << curr_psnr;

		// Training stop ?
		if (testbed.m_training_step >= max_epoch || curr_psnr >= max_psnr 
			|| (std::time(nullptr) - start_time) >= max_time) {
			break;
		}
	}

	if (save_model_flag) {
	    fs::path filename = get(save_model_flag);
	    save_model(testbed, filename);
	}

	if (save_mesh_flag) {
	    fs::path filename = get(save_mesh_flag);
	    save_mesh(testbed, filename);
	}

	if (save_point_flag) {
	    fs::path filename = get(save_point_flag);
	    save_point(testbed, filename);
	}

	return 0;
}

NGP_NAMESPACE_END

#ifdef _WIN32
int wmain(int argc, wchar_t* argv[]) {
	SetConsoleOutputCP(CP_UTF8);
#else
int main(int argc, char* argv[]) {
#endif
	try {
		std::vector<std::string> arguments;
		for (int i = 0; i < argc; ++i) {
#ifdef _WIN32
			arguments.emplace_back(ngp::utf16_to_utf8(argv[i]));
#else
			arguments.emplace_back(argv[i]);
#endif
		}
		if (argc == 1) {
			arguments.emplace_back("--help");
		}

		return ngp::main_func(arguments);
	} catch (const exception& e) {
		tlog::error() << fmt::format("Uncaught exception: {}", e.what());
		return 1;
	}
}
