#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   main.cu
 *  @author Thomas Müller, NVIDIA
 */

#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

#include <args/args.hxx>

#include <filesystem/path.h>

using namespace args;
using namespace ngp;
using namespace std;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

#define DEF_GUI_WIDTH "1920"
#define DEF_GUI_HEIGHT "1080"
#define DEF_MAX_TIME "36000" // 10 hours
#define DEF_MAX_PSNR "50.00"
#define DEF_MAX_EPOCH "1000000"

#define file_like(filename, likes) equals_case_insensitive(filename.extension(), likes)


inline float psnr(float x)
{
	return -10.f*logf(x)/logf(10.f);
}

void load_model(Testbed &testbed, fs::path &filename)
{
    if (! filename.exists()) {
		tlog::warning() << "Model file '" << filename.str() << "' does not exist";
		return;
    }

    if (! file_like(filename, "msgpack")) {
		tlog::warning() << "Model should be '*.msgpack' file";
		return;
    }

	testbed.load_snapshot(filename);
}


int main_func(const std::vector<std::string>& arguments) {
	ArgumentParser parser{
		"Instant Neural Graphics Primitives Version " NGP_VERSION,
		"",
	};

	HelpFlag help_flag{
		parser,
		"HELP",
		"Display this help.",
		{'h', "help"},
	};

	Flag version_flag{
		parser,
		"VERSION",
		"Display version.",
		{'v', "version"},
	};

	Flag no_gui_flag{
		parser,
		"NO_GUI",
		"Disable GUI.",
		{"no_gui"},
	};

	ValueFlag<uint32_t> width_flag{
		parser,
		DEF_GUI_WIDTH,
		"GUI width.",
		{"width"},
	};

	ValueFlag<uint32_t> height_flag{
		parser,
		DEF_GUI_HEIGHT,
		"GUI height.",
		{"height"},
	};

#ifdef NGP_GUI
	Flag vr_flag{
		parser,
		"VR",
		"Enable VR",
		{"vr"}
	};
#endif

	ValueFlag<string> load_config_flag{
		parser,
		"CONFIG_FILE",
		"Load network config from *.json file.",
		{"load_config"},
	};

	ValueFlag<string> load_model_flag{
		parser,
		"MODEL_FILE",
		"Load model from *.msgpack file.",
		{"load_model"},
	};

	ValueFlag<string> load_data_flag{
		parser,
		"DATASET_NAME",
		"Load training data from dataset (Folder for NeRF, *.obj/*.stl for SDF, *.nvdb for volume, others for image ).",
		{"load_data"},
	};

	ValueFlag<string> output_dir_flag{
		parser,
		"output",
		"Set output directory.",
		{"output"},
	};


	Flag save_model_flag{
		parser,
		"SAVE_MODEL",
		"Save model to output/model.msgpack.",
		{"save_model"},
	};

	Flag save_mesh_flag{
		parser,
		"SAVE_MESH",
		"Save mesh to output/mesh.obj for NeRF or SDF.",
		{"save_mesh"},
	};

	Flag save_images_flag{
		parser,
		"SAVE_IMAGES",
		"Render images/depth to output for NeRF.",
		{"save_images"},
	};

	ValueFlag<float> save_points_flag{
		parser,
		"S%",
		"Save point cloud (sample S%) to output/3d_pc.ply for NeRF.",
		{"save_points"},
	};

	Flag no_train_flag{
		parser,
		"NO_TRAIN",
		"Disable training.",
		{"no_train"},
	};

	ValueFlag<int32_t> max_epoch_flag{
		parser,
		DEF_MAX_EPOCH,
		"Training stop if epoch >= max_epoch.",
		{"max_epoch"},
	};

	ValueFlag<int32_t> max_time_flag{
		parser,
		DEF_MAX_TIME,
		"Training stop if time >= max_time seconds.",
		{"max_time"},
	};

	ValueFlag<float> max_psnr_flag{
		parser,
		DEF_MAX_PSNR,
		"Training stop if PSNR >= max_psnr.",
		{"max_psnr"},
	};

	PositionalList<string> files{
		parser,
		"files",
		"Files to be loaded. Can be a dataset, network config, snapshot, camera path, or a combination of those.",
	};

	// Parse command line arguments and react to parsing
	// errors using exceptions.
	try {
		if (arguments.empty()) {
			tlog::error() << "Argument number must be > 0.";
			return -3;
		}

		parser.Prog(arguments.front());
		parser.ParseArgs(begin(arguments) + 1, end(arguments));
	} catch (const Help&) {
		cout << parser;
		return 0;
	} catch (const ParseError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -1;
	} catch (const ValidationError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -2;
	}

	if (version_flag) {
		tlog::none() << "Instant Neural Graphics Primitives v" NGP_VERSION;
		return 0;
	}

	// Start ...
	Testbed testbed;
	
	for (auto file : get(files)) {
		testbed.load_file(file);
	}

	if (load_data_flag) {
		testbed.load_training_data(get(load_data_flag));
	}

	if (load_config_flag) {
		testbed.reload_network_from_file(get(load_config_flag));
	}

	if (load_model_flag) {
	    fs::path filename = get(load_model_flag);
	    load_model(testbed, filename);
	}

	testbed.m_train = !no_train_flag;

#ifdef NGP_GUI
	bool gui = !no_gui_flag;
#else
	bool gui = false;
#endif
	if (! testbed.m_train) // set no-gui without training
		gui = false;

	if (gui) {
		testbed.init_window(width_flag ? get(width_flag) : atoi(DEF_GUI_WIDTH),
			height_flag ? get(height_flag) : atoi(DEF_GUI_HEIGHT));
	}

#ifdef NGP_GUI
	if (vr_flag) {
		testbed.init_vr();
	}
#endif

	// Render/training loop
	float curr_psnr = 0.0f;
	std::time_t start_time = std::time(nullptr);
	float max_psnr = (max_psnr_flag)? get(max_psnr_flag) : atof(DEF_MAX_PSNR);
	uint32_t max_time = (max_time_flag)? get(max_time_flag) : atoi(DEF_MAX_TIME);
	uint32_t max_epoch = (max_epoch_flag)? get(max_epoch_flag) : atoi(DEF_MAX_EPOCH);

	testbed.m_training_step = 0;
	while (testbed.m_train && testbed.frame()) {
		if (testbed.m_training_step % 100 != 0)
			continue;

		curr_psnr = psnr(testbed.m_loss_scalar.val());
		tlog::info() << "iteration=" << testbed.m_training_step 
				<< " loss=" << testbed.m_loss_scalar.val()
				<< " psnr=" << curr_psnr
				<< " memory=" << testbed.gpu_memory_used();

		// Training stop ?
		if (testbed.m_training_step >= max_epoch || curr_psnr >= max_psnr 
			|| (std::time(nullptr) - start_time) >= max_time) {
			break;
		}
	}

	fs::path output_dir = "output";
	if (output_dir_flag || save_model_flag || save_images_flag || save_mesh_flag || save_points_flag) {
		output_dir = output_dir_flag ? get(output_dir_flag) : "output";
		if (! output_dir.is_directory()) {
			fs::create_directory(output_dir);
		}
	}

	if (save_model_flag) {
	    fs::path filename = output_dir/"model.msgpack";
		testbed.save_snapshot(filename, false /*include_optimizer_state*/);
	}

	if (save_mesh_flag) {
		if (testbed.m_testbed_mode != ETestbedMode::Nerf && 
			testbed.m_testbed_mode != ETestbedMode::Sdf) {
			tlog::warning() << "Save mesh only for NeRF or SDF.";
		} else {
		    fs::path filename = output_dir/"mesh.obj";
			float thresh = (testbed.m_testbed_mode == ETestbedMode::Nerf)? 2.5f : 0.0f;
			testbed.compute_and_save_marching_cubes_mesh(filename.str().c_str(),
				Eigen::Vector3i{256, 256, 256}, {} /*BoundingBox*/, thresh, false /*uv_flag*/);
		}
	}

	if (save_images_flag) {
		if (testbed.m_testbed_mode != ETestbedMode::Nerf) {
			tlog::warning() << "Render images/depth only for NeRF.";
		} else {
			testbed.save_nerf_images(output_dir);
		}
	}

	if (save_points_flag) {
		if (testbed.m_testbed_mode != ETestbedMode::Nerf) {
			tlog::warning() << "Save point cloud only for NeRF.";
		} else {
			float ratio = get(save_points_flag);
		    fs::path filename = output_dir/"3d_pc.ply";
			if (ratio < 0.0f)
				ratio = 100.0f;
			if (ratio > 100.0f)
				ratio = 100.0f;

			testbed.save_nerf_points(ratio, filename.str().c_str());
		}
	}

	return 0;
}

NGP_NAMESPACE_END

#ifdef _WIN32
int wmain(int argc, wchar_t* argv[]) {
	SetConsoleOutputCP(CP_UTF8);
#else
int main(int argc, char* argv[]) {
#endif
	try {
		std::vector<std::string> arguments;
		for (int i = 0; i < argc; ++i) {
#ifdef _WIN32
			arguments.emplace_back(ngp::utf16_to_utf8(argv[i]));
#else
			arguments.emplace_back(argv[i]);
#endif
		}
		if (argc == 1) {
			arguments.emplace_back("--help");
		}

		return ngp::main_func(arguments);
	} catch (const exception& e) {
		tlog::error() << fmt::format("Uncaught exception: {}", e.what());
		return 1;
	}
}
